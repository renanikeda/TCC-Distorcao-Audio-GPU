#include "hip/hip_runtime.h"
//Bibliotecas
////////////////////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "sndfile.h"
#include <math.h>
#include <hip/hip_runtime_api.h> 
#include <time.h> 
////////////////////////////////////////////////////////////////////////////////////////////////

//Defines ajustaveis
////////////////////////////////////////////////////////////////////////////////////////////////
#define L 30
#define N_interpol 6
#define N_decimacao 6
#define N_aliasing 7
#define N_operacoes_case0 125//Numero de filtragens no kernel de Interpolacao
#define N_operacoes_case1 125 //Numero de filtragens no kernel de decimacao
#define N_operacoes_case2  125//Numero de filtragens no kernel de distorcao
#define N_filtragens_interpol 4 //Numero de filtragens pelo filtro de interpolacao
#define N_filtragens_decimacao 4//Numero de filtragens pelo filtro de decimacao
#define N_estagios 16 //Numero de estagios (incluindo Interpolacao e Decimacao)
#define N_buffer 10000
////////////////////////////////////////////////////////////////////////////////////////////////

//Defines em função
////////////////////////////////////////////////////////////////////////////////////////////////
#define N_eventos (N_estagios+1)
#define N_streams (N_estagios+2)
#define N_estagios_menos_1 (N_estagios-1)
#define N_estagios_menos_2 (N_estagios-2)
#define N_aliasing_mais_1 (N_aliasing+1)
#define N_interpol_mais_1 (N_interpol+1)
#define N_decimacao_mais_1 (N_decimacao+1)
#define offset_decimacao (N_operacoes_case0+N_operacoes_case2*(N_estagios_menos_2))
#define N_distorcao (N_operacoes_case0+N_operacoes_case1+N_operacoes_case2*(N_estagios_menos_2))
#define N_eventos_mais_um (N_eventos+1)
/////////////////////////////////////////////////////////////////////////////////////////////////

//Kernel de Interpolacao (N_operacoes_case0+N_filtragens_interpol)
__global__ void Interpolacao(
    double* amostra_in, double* amostra_out,
    double* entrada_interpol, double* saida_interpol,
    double* entrada_aliasing, double* saida_aliasing,
    double *a_interpol, double *b_interpol,
    double *a_aliasing, double *b_aliasing,
    double grau_distorcao)

{    
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_interpolacao[N_interpol_mais_1];
    double saida_multiplicada_interpolacao[N_interpol];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];

    posicao[gid]=-gid;
    amostra_out[gid]=amostra_in[0];

    while(posicao[L-1]<N_operacoes_case0+N_filtragens_interpol)
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

    
        if(posicao[gid]>-1 && posicao[gid]<N_filtragens_interpol) //Interpolacao
        {
            for(int i=N_interpol; i>0; i--)
            {
                entrada_interpol[i+(N_interpol+1)*posicao[gid]]=entrada_interpol[i-1+(N_interpol+1)*posicao[gid]];
            }
            entrada_interpol[(N_interpol+1)*posicao[gid]]=amostra_out[gid];

            for(int i=0; i<N_interpol_mais_1; i++)
            {
                entrada_multiplicada_interpolacao[i]=entrada_interpol[i+(N_interpol+1)*posicao[gid]]*b_interpol[i];
                soma_entrada[gid]+=entrada_multiplicada_interpolacao[i];
            }
            for(int i=0; i<N_interpol; i++)
            {
                saida_multiplicada_interpolacao[i]=saida_interpol[i+(N_interpol)*posicao[gid]]*a_interpol[i];
                soma_saida[gid]+=saida_multiplicada_interpolacao[i];
            }

            amostra_out[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_interpol-1; i>0; i--)
            {
                saida_interpol[i+(N_interpol)*posicao[gid]]=saida_interpol[i-1+(N_interpol)*posicao[gid]];
            }
            saida_interpol[(N_interpol)*posicao[gid]]=amostra_out[gid];

            if(posicao[gid]==N_filtragens_interpol-1)
            {
                if(amostra_out[gid]>1)
                {
                    amostra_out[gid]=1;
                }
                else if(amostra_out[gid]<-1)
                {
                    amostra_out[gid]=-1;
                }
            }   
        }
        else if(posicao[gid]>(N_filtragens_interpol-1) && posicao[gid]<N_operacoes_case0+(N_filtragens_interpol)) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[0+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_out[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]-N_filtragens_interpol)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_out[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]-N_filtragens_interpol)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]-N_filtragens_interpol)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]-N_filtragens_interpol)*N_aliasing]=amostra_out[gid];

        }

        posicao[gid]+=1;

        __syncthreads();        
        
            
    }

}

//Kernel de Distorcao (N_operacoes_case2)
__global__ void Distorcao(
    double* amostra_in, double* amostra_out,
    double* entrada_aliasing, double* saida_aliasing,
    double *a_aliasing, double *b_aliasing,
    double grau_distorcao, int estagio
)
{    
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];
   

    posicao[gid]=-gid;

    while(posicao[L-1]<N_operacoes_case2)
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

      
        if(posicao[gid]>-1 && posicao[gid]<N_operacoes_case2) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[0+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_in[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]=amostra_in[gid];
        }
                 
    
        posicao[gid]+=1;

        __syncthreads();
    }
}

//Kernel de Decimacao (N_operacoes_case1-1 distorcoes)
__global__ void Decimacao(
    double* amostra_in, double* amostra_out,
    double* entrada_aliasing, double* saida_aliasing,
    double* entrada_decimacao, double* saida_decimacao,
    double *a_aliasing, double *b_aliasing,
    double *a_decimacao, double *b_decimacao,
    double grau_distorcao
)
{    
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_decimacao[N_decimacao_mais_1];
    double saida_multiplicada_decimacao[N_decimacao];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];

    posicao[gid]=-gid;

    while(posicao[L-1]<N_operacoes_case1+(N_filtragens_decimacao))
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

        if(posicao[gid]>N_operacoes_case1-1 && posicao[gid]<N_operacoes_case1+(N_filtragens_decimacao)) //Decimacao
        {
            for(int i=N_decimacao; i>0; i--)
            {
                entrada_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]=entrada_decimacao[i-1+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)];
            }
            entrada_decimacao[(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]=amostra_in[gid];

            for(int i=0; i<N_decimacao_mais_1; i++)
            {
                entrada_multiplicada_decimacao[i]=entrada_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]*b_decimacao[i];
                soma_entrada[gid]+=entrada_multiplicada_decimacao[i];
            }
            for(int i=0; i<N_decimacao; i++)
            {
                saida_multiplicada_decimacao[i]=saida_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao)]*a_decimacao[i];
                soma_saida[gid]+=saida_multiplicada_decimacao[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_decimacao-1; i>0; i--)
            {
                saida_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao)]=saida_decimacao[i-1+(posicao[gid]-N_operacoes_case1)*(N_decimacao)];
            }
            saida_decimacao[(posicao[gid]-N_operacoes_case1)*(N_decimacao)]=amostra_in[gid];

            if(gid==0 && posicao[gid]==N_operacoes_case1+(N_filtragens_decimacao)-1)
            {
                amostra_out[0]=amostra_in[gid];
            }

        }
        else if(posicao[gid]>-1 && posicao[gid]<N_operacoes_case1) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_in[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]+offset_decimacao)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]+offset_decimacao)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]+offset_decimacao)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]+offset_decimacao)*N_aliasing]=amostra_in[gid];
        }

       
        posicao[gid]+=1;

        __syncthreads();
    }
    
}

//Kernel para verificar uma variavel do device 
__global__ void Verifica_buffer(double *bufferd, int i, int nome)
{
    if(nome == 0)
        printf("H/D buffer[%d] = %e\n", i, bufferd[i]);
    else
        printf("D/H buffer[%d] = %e\n", i, bufferd[i]);
}

//Kernel para verificar um vetor do device
__global__ void Verifica_vetor(double *bufferd, int N)
{
    for (int i=0; i<N;i++)
    {
        printf("vetor[%d]=%e\n", i, bufferd[i]);
    }
}

int main(){

    printf("N_distorcao: %d\n", N_distorcao);
    printf("Definindo arquivo de entrada e saída: \n");
    
    //Arquivo de entrada e saida
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Definições do arquivo de leitura e saida
    SNDFILE *file_in,*file_out ;   //Arquivo de entrada e saída
    SF_INFO sfinfo_in,sfinfo_out ; //Arquivo de informações de entrada e saída
    sfinfo_in.format = 0;          //Documentação do libsnd manda fazer isso para arquivos de leitura
    file_in = sf_open ("sweep_48khz_5s.wav", SFM_READ, &sfinfo_in); //Determina o arquivo de entrada
    sfinfo_out=sfinfo_in;
    file_out = sf_open ("sweep_48khz_5s_2000_0.045.wav", SFM_WRITE, &sfinfo_out); //Determina o arquivo de saída
    sf_command (file_out, SFC_SET_CLIPPING, NULL, SF_TRUE) ;
    printf("Informações sobre o arquivo de entrada: \n");
    printf("Taxa de amostragem = %d , Frames = % d , Canais = % d \n" , (int) sfinfo_in.samplerate,(int) sfinfo_in.frames,(int)sfinfo_in.channels); 
    //Mostra algumas caratersiticas do arquivo de entrada
    int read_count = 1;
    ////////////////////////////////////////////////////////////////////////////////////////////////

    printf("Alocando memória: \n");

    //Definicao das variaveis e alocacao de memoria
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Definições do Projeto
    const double grau_distorcao=0.045; //Grau de distorcao do atan
    const int N_amostras= sfinfo_in.frames; //Numero de amostras do arquivo de entrada
    //const int N_amostras = 1000; //Para verficacao da timeline

    //Variaveis da CPU
    double *amostra_in_host; //Amostra para a entrada no processamento feito na GPU
    double *amostra_out_host; //Amostra para a saída no processamento feito na GPU

    //Alocação das Variaveis da CPU
    hipHostAlloc((void**)&amostra_in_host, sizeof(double)  *N_buffer,hipHostMallocDefault);
    hipHostAlloc((void**)&amostra_out_host, sizeof(double)  *N_buffer,hipHostMallocDefault);

    //Definição dos coeficientes do filtro na CPU
    double a_interpol_host[N_interpol] = {
        #include "a_interpol_30_final.txt"
    };
    double b_interpol_host[N_interpol_mais_1] = {
        #include "b_interpol_30_final.txt"
    };
    double a_decimacao_host[N_decimacao] = {
        #include "a_decimacao_30_final.txt"
    };
    double b_decimacao_host[N_decimacao_mais_1] = {
        #include "b_decimacao_30_final.txt"
    };
    double a_aliasing_host[N_aliasing] = {
        #include "a_aliasing_30_final.txt"
    };
    double b_aliasing_host[N_aliasing_mais_1] = {
        #include "b_aliasing_30_final.txt"
    };

    //Variaveis da GPU
    double *buffer_device_in; //Buffer do devide para leitura do arquivo
    double *buffer_device_out; //Buffer do para escrever amostras processadas
    double *amostra_estagio; //Buffer intermediario das distorcoes
    double *a_interpol; //Vetor dos coeficientes recursivos do filtro de interpolação 
    double *b_interpol; //Vetor dos coeficientes relacionados a entrada do filtro de interpolação
    double *a_decimacao; //Vetor dos coeficientes recursivos do filtro de decimacao
    double *b_decimacao; //Vetor dos coeficientes relacionados a entrada do filtro de decimação
    double *a_aliasing; //Vetor dos coeficientes recursivos do filtro pós distorção 
    double *b_aliasing; //Vetor dos coeficientes relacionados a entrada do filtro pós distorção
    double *entrada_interpol; //Vetor de entrada na interpolação
    double *saida_interpol; //Vetor de saída na interpolação
    double *entrada_decimacao; //Vetor de entrada na decimação
    double *saida_decimacao; //Vetor de saída na decimação
    double *entrada_aliasing; //Vetor de entrada no filtro pós distorção
    double *saida_aliasing; //Vetor de saída no filtro pós distorção


    //Alocação das Variaveis da GPU
    hipMalloc((void**)&amostra_estagio, sizeof(double)* L*N_buffer);
    hipMalloc((void**)&buffer_device_in, sizeof(double)*N_buffer);
    hipMalloc((void**)&buffer_device_out, sizeof(double)*N_buffer);
    hipMalloc((void**)&a_interpol, sizeof(double) * N_interpol);
    hipMalloc((void**)&b_interpol, sizeof(double) * (N_interpol_mais_1));
    hipMalloc((void**)&a_decimacao, sizeof(double) * N_decimacao);
    hipMalloc((void**)&b_decimacao, sizeof(double) * (N_decimacao_mais_1));
    hipMalloc((void**)&a_aliasing, sizeof(double) * N_aliasing);
    hipMalloc((void**)&b_aliasing, sizeof(double) * (N_aliasing_mais_1));
    hipMalloc((void**)&saida_interpol, sizeof(double) * (N_filtragens_interpol)*N_interpol);
    hipMalloc((void**)&entrada_interpol, sizeof(double) * (N_filtragens_interpol)*(N_interpol_mais_1));
    hipMalloc((void**)&saida_decimacao, sizeof(double) * (N_filtragens_decimacao)*N_decimacao);
    hipMalloc((void**)&entrada_decimacao, sizeof(double) * (N_filtragens_decimacao)*(N_decimacao_mais_1));
    hipMalloc((void**)&entrada_aliasing, sizeof(double) * N_distorcao*(N_aliasing_mais_1));
    hipMalloc((void**)&saida_aliasing, sizeof(double) * N_distorcao*N_aliasing);
    

    //Definição dos coeficientes dos filtros na GPU;
    hipMemcpy(a_interpol, a_interpol_host, sizeof(double)*N_interpol, hipMemcpyHostToDevice);
    hipMemcpy(b_interpol, b_interpol_host, sizeof(double)*(N_interpol_mais_1), hipMemcpyHostToDevice);
    hipMemcpy(a_decimacao, a_decimacao_host, sizeof(double)*N_decimacao, hipMemcpyHostToDevice);
    hipMemcpy(b_decimacao, b_decimacao_host, sizeof(double)*(N_decimacao_mais_1), hipMemcpyHostToDevice);
    hipMemcpy(a_aliasing, a_aliasing_host, sizeof(double)*N_aliasing, hipMemcpyHostToDevice);
    hipMemcpy(b_aliasing, b_aliasing_host, sizeof(double)*(N_aliasing_mais_1), hipMemcpyHostToDevice);


    //Definindo filtros com condicoes iniciais nulas
    hipMemset((void**)&saida_interpol, 0, sizeof(double)*(N_filtragens_interpol)*N_interpol); 
    hipMemset((void**)&entrada_interpol, 0, sizeof(double)*(N_filtragens_interpol)*(N_interpol_mais_1)); 
    hipMemset((void**)&saida_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*N_decimacao);
    hipMemset((void**)&entrada_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*(N_decimacao_mais_1));
    hipMemset((void**)&saida_aliasing, 0, sizeof(double)*N_aliasing*N_distorcao);
    hipMemset((void**)&entrada_aliasing, 0, sizeof(double)*(N_aliasing_mais_1)*N_distorcao);
    hipMemset((void**)&buffer_device_out, 2, sizeof(double)*N_buffer);
    hipMemset((void**)&buffer_device_in, 0, sizeof(double)*N_buffer);
    
    //Definindo Streams
    hipStream_t estagio[N_streams];
    for(int i=0; i<(N_streams); i++)
    {
        hipStreamCreate(&estagio[i]);
    }

    //Definindo Eventos
    hipEvent_t *fim_estagio;
    fim_estagio=(hipEvent_t*)malloc(sizeof(hipEvent_t)* (N_buffer+1)*(N_eventos_mais_um));
    for(int i=0; i<(N_buffer+1)*(N_eventos_mais_um); i++)
    {
        hipEventCreateWithFlags(&fim_estagio[i],hipEventDisableTiming);
    }

    //Variavel de confirmacao de eventos (mem dinamica)
    // int* event_ok;
    // event_ok=(int*)calloc((N_buffer+1)*(N_eventos_mais_um),sizeof(int));

    //Variavel de confirmacao de eventos (mem estatica)
    int event_ok[(N_buffer+1)*(N_eventos_mais_um)];
    for(int i=0; i<(N_buffer+1)*(N_eventos_mais_um) ; i++)
    {
        event_ok[i]=0;
    }

    //Variaveis de medida de tempo
    clock_t inicio_total;
	clock_t fim_total;
    double soma_total = 0;
    int pct=0;
    int amostras_enviadas=0;

    //Contadores de cada estagio
    int amostras_processadas[N_streams];
    int amostras_processadas_totais=0;
    for(int i=0; i<N_streams; i++)
    {
        amostras_processadas[i]=0;
    }
    
    //Arquivo de log
    FILE *fp = fopen("log_TCC_jerson_query_debug.txt","w");
 
    ////////////////////////////////////////////////////////////////////////////////////////////////

    printf("Começando processamento: \n");

    inicio_total = clock();
    while (amostras_processadas_totais< N_amostras)
    {
        if(amostras_processadas[0]< N_buffer && amostras_processadas[0]==amostras_processadas[1])
        {
            read_count = (int) sf_read_double (file_in, &amostra_in_host[amostras_processadas[0]], 1);
            hipStreamSynchronize(estagio[1]);
            if(read_count==1)
            {
                hipMemcpyAsync(&buffer_device_in[amostras_processadas[0]], &amostra_in_host[amostras_processadas[0]], sizeof(double), hipMemcpyHostToDevice,estagio[0]);
                hipEventRecord(fim_estagio[(N_eventos_mais_um)*amostras_processadas[0]],estagio[0]);
                event_ok[(N_eventos_mais_um)*amostras_processadas[0]]=1;
                amostras_processadas[0]++;
                amostras_enviadas++;
            }
        }

        if(hipEventQuery(fim_estagio[(amostras_processadas[1])*(N_eventos_mais_um)])==hipSuccess && (event_ok[(amostras_processadas[1])*(N_eventos_mais_um)]==1))
        {
            event_ok[(amostras_processadas[1])*(N_eventos_mais_um)]=0;

            Interpolacao<<<1,L,0,estagio[1]>>>(
                &buffer_device_in[amostras_processadas[1]],&amostra_estagio[L*amostras_processadas[1]],
                entrada_interpol,saida_interpol,
                entrada_aliasing,saida_aliasing,
                a_interpol, b_interpol,
                a_aliasing, b_aliasing,
                grau_distorcao);
            hipEventRecord(fim_estagio[1+amostras_processadas[1]*(N_eventos_mais_um)],estagio[1]);
            // Verifica_vetor<<<1,1,0,estagio[1]>>>(&amostra_estagio[L*amostras_processadas[1]],L);
            // hipStreamSynchronize(estagio[1]);
            event_ok[(N_eventos_mais_um)*amostras_processadas[1]+1]=1;
            amostras_processadas[1]++;
            fprintf(fp,"Interpolacao amostras_processadas[1]=%d \n", amostras_processadas[1]);
        }

        for(int z=1; z<N_estagios_menos_1;z++)
        {
            if(hipEventQuery(fim_estagio[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)])==hipSuccess && event_ok[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)]==1)
            {
                event_ok[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)]=0;

                Distorcao<<<1,L,0,estagio[(z+1)]>>>(
                    &amostra_estagio[L*amostras_processadas[(z+1)]],&amostra_estagio[L*amostras_processadas[(z+1)]],
                    entrada_aliasing,saida_aliasing,
                    a_aliasing, b_aliasing,
                    grau_distorcao,z);
                hipEventRecord(fim_estagio[z+1+amostras_processadas[(z+1)]*(N_eventos_mais_um)],estagio[(z+1)]);
                event_ok[z+1+(N_eventos_mais_um)*amostras_processadas[(z+1)]]=1;
                amostras_processadas[(z+1)]++;
                fprintf(fp,"Distorcao amostras_processadas[2]=%d \n", amostras_processadas[(z+1)]);
            }
        } 
       
      
        if(hipEventQuery(fim_estagio[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)])==hipSuccess && event_ok[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]==1)
        {
            event_ok[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]=0;

            Decimacao<<<1,L,0,estagio[(N_estagios)]>>>(
                &amostra_estagio[L*amostras_processadas[(N_estagios)]],&buffer_device_out[amostras_processadas[(N_estagios)]],
                entrada_aliasing,saida_aliasing,
                entrada_decimacao, saida_decimacao,
                a_aliasing, b_aliasing,
                a_decimacao, b_decimacao,
                grau_distorcao);
            hipEventRecord(fim_estagio[(N_estagios)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)],estagio[(N_estagios)]);
            event_ok[N_estagios+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]=1;
            amostras_processadas[(N_estagios)]++;
            fprintf(fp,"Decimacao amostras_processadas[(N_estagios)]=%d \n", amostras_processadas[(N_estagios)]);
        }
        
        if(hipEventQuery(fim_estagio[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]])==hipSuccess && event_ok[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]==1)
        {
            event_ok[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]=0;

            if(amostras_processadas_totais % ((int)sfinfo_in.frames / 10) == 0)
            {
                printf("Porcentagem do processamento = %d%c \n", pct,37);
                pct+=10;
            }
            hipMemcpyAsync(&amostra_out_host[amostras_processadas[(N_eventos)]], &buffer_device_out[amostras_processadas[(N_eventos)]], sizeof(double), hipMemcpyDeviceToHost,estagio[(N_eventos)]);
            hipEventRecord(fim_estagio[(N_eventos)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]],estagio[(N_eventos)]);
            event_ok[(N_eventos)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]=1;
            amostras_processadas[(N_eventos)]++;
            amostras_processadas_totais++;
        }


        if(amostras_processadas[(N_eventos)] == (N_buffer) && amostras_processadas[0]== (N_buffer) )
        {
            if(hipEventQuery(fim_estagio[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)])==hipSuccess && event_ok[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)]==1)
            {
                sf_write_double (file_out, amostra_out_host, N_buffer ) ;
                event_ok[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)]=0;
                for(int k=0; k<N_eventos_mais_um;k++)
                {
                    amostras_processadas[k]=0;
                }
            }
        }

    }   

    hipDeviceSynchronize();
    
    ////////////////////////////////////////////////////////////////////////////////////////////////

    printf("Fim do processamento: \n");

    fim_total = clock();

    soma_total += ((double)(fim_total - inicio_total)) / CLOCKS_PER_SEC;

    sf_write_double (file_out, amostra_out_host, amostras_processadas[(N_eventos)] ) ;

    //Prints finais 
    ////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Nro de blocos processados.....: %d\n", amostras_processadas_totais);
    printf("\n");
    printf("Tempo Total \n"); 
    printf("Tempo total de processamento com tranf de memoria [s]...: %f \n", soma_total);

    printf("\n");
    printf("Porcentagens do tempo total de processamento \n");
    printf("PCT do tempo de processamento [s]...: %f \n", 100*(soma_total/soma_total));
  

    printf("Liberando a memória alocada: \n");

   
    //Liberando a memória alocada
    ////////////////////////////////////////////////////////////////////////////////////////////////
    // free(event_ok);

    
    hipFree(amostra_in_host);
    hipFree(amostra_out_host);

    hipFree(buffer_device_in);
    hipFree(buffer_device_out);
    hipFree(amostra_estagio);

    hipFree(a_interpol);
    hipFree(b_interpol);
    hipFree(a_decimacao);
    hipFree(b_decimacao);
    hipFree(a_aliasing);
    hipFree(b_aliasing);
    hipFree(entrada_interpol);
    hipFree(saida_interpol);
    hipFree(entrada_decimacao);
    hipFree(saida_decimacao);
    hipFree(entrada_aliasing);
    hipFree(saida_aliasing);

    for(int i=0; i<(N_buffer+1)*(N_eventos); i++)
    {
        hipEventDestroy(fim_estagio[i]);
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////

     printf("Fim: \n");
 }