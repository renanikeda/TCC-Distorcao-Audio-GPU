#include "hip/hip_runtime.h"
//Bibliotecas
////////////////////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "sndfile.h"
#include <math.h>
#include <hip/hip_runtime_api.h> 
#include <time.h> 
////////////////////////////////////////////////////////////////////////////////////////////////

//Defines ajustaveis
////////////////////////////////////////////////////////////////////////////////////////////////
#define L 30
#define N_interpol 7
#define N_decimacao 7
#define N_aliasing 7
#define N_max_estagios 100

#define N_filtragens_interpol 3    //Numero de filtragens pelo filtro de interpolacao
#define N_filtragens_decimacao 3   //Numero de filtragens pelo filtro de decimacao
#define N_buffer 10000
////////////////////////////////////////////////////////////////////////////////////////////////

//Defines em função
////////////////////////////////////////////////////////////////////////////////////////////////
#define N_aliasing_mais_1 (N_aliasing+1)
#define N_interpol_mais_1 (N_interpol+1)
#define N_decimacao_mais_1 (N_decimacao+1)
#define N_distorcao_max 500
/////////////////////////////////////////////////////////////////////////////////////////////////

//Kernel de Interpolacao (N_operacoes_case0-1 distorcoes)
__global__ void Interpolacao(
    double* amostra_in, double* amostra_out,
    double* entrada_interpol, double* saida_interpol,
    double* entrada_aliasing, double* saida_aliasing,
    double *a_interpol, double *b_interpol,
    double *a_aliasing, double *b_aliasing,
    double grau_distorcao, int N_operacoes_case0)

{
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_interpolacao[N_interpol_mais_1];
    double saida_multiplicada_interpolacao[N_interpol];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];

    posicao[gid]=-gid;
    amostra_out[gid]=amostra_in[0];

    while(posicao[L-1]<N_operacoes_case0+N_filtragens_interpol)
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

    
        if(posicao[gid]>-1 && posicao[gid]<N_filtragens_interpol) //Interpolacao
        {
            for(int i=N_interpol; i>0; i--)
            {
                entrada_interpol[i+(N_interpol+1)*posicao[gid]]=entrada_interpol[i-1+(N_interpol+1)*posicao[gid]];
            }
            entrada_interpol[(N_interpol+1)*posicao[gid]]=amostra_out[gid];

            for(int i=0; i<N_interpol_mais_1; i++)
            {
                entrada_multiplicada_interpolacao[i]=entrada_interpol[i+(N_interpol+1)*posicao[gid]]*b_interpol[i];
                soma_entrada[gid]+=entrada_multiplicada_interpolacao[i];
            }
            for(int i=0; i<N_interpol; i++)
            {
                saida_multiplicada_interpolacao[i]=saida_interpol[i+(N_interpol)*posicao[gid]]*a_interpol[i];
                soma_saida[gid]+=saida_multiplicada_interpolacao[i];
            }

            amostra_out[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_interpol-1; i>0; i--)
            {
                saida_interpol[i+(N_interpol)*posicao[gid]]=saida_interpol[i-1+(N_interpol)*posicao[gid]];
            }
            saida_interpol[(N_interpol)*posicao[gid]]=amostra_out[gid];

            if(posicao[gid]==N_filtragens_interpol-1)
            {
                if(amostra_out[gid]>1)
                {
                    amostra_out[gid]=1;
                }
                else if(amostra_out[gid]<-1)
                {
                    amostra_out[gid]=-1;
                }
            }   
        }
        else if(posicao[gid]>(N_filtragens_interpol-1) && posicao[gid]<N_operacoes_case0+(N_filtragens_interpol)) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[0+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_out[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]-N_filtragens_interpol)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]-N_filtragens_interpol)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_out[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]-N_filtragens_interpol)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]-N_filtragens_interpol)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]-N_filtragens_interpol)*N_aliasing]=amostra_out[gid];

        }

        posicao[gid]+=1;

        __syncthreads();
    }
}

//Kernel de Distorcao (N_operacoes_case2)
__global__ void Distorcao(
    double* amostra_in, double* amostra_out,
    double* entrada_aliasing, double* saida_aliasing,
    double *a_aliasing, double *b_aliasing,
    double grau_distorcao, int estagio, int N_operacoes_case0, int N_operacoes_case2
)
{    
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];

    posicao[gid]=-gid;

    while(posicao[L-1]<N_operacoes_case2)
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

        if(posicao[gid]>-1 && posicao[gid]<N_operacoes_case2) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[0+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_in[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]+N_operacoes_case0+(estagio-1)*N_operacoes_case2)*N_aliasing]=amostra_in[gid];
        }

        posicao[gid]+=1;

        __syncthreads();
    }
}

//Kernel de Decimacao (N_operacoes_case1-1 distorcoes)
__global__ void Decimacao(
    double* amostra_in, double* amostra_out,
    double* entrada_aliasing, double* saida_aliasing,
    double* entrada_decimacao, double* saida_decimacao,
    double *a_aliasing, double *b_aliasing,
    double *a_decimacao, double *b_decimacao,
    double grau_distorcao, int N_operacoes_case0, int N_operacoes_case2, int N_operacoes_case1, int N_estagios
)
{    
    int gid = blockIdx.x*blockDim.x+threadIdx.x;
    __shared__ int posicao[L];
    __shared__ double soma_entrada[L];
    __shared__ double soma_saida[L];
    double entrada_multiplicada_decimacao[N_decimacao_mais_1];
    double saida_multiplicada_decimacao[N_decimacao];
    double entrada_multiplicada_aliasing[N_aliasing_mais_1];
    double saida_multiplicada_aliasing[N_aliasing];

    int offset_decimacao = N_operacoes_case0+N_operacoes_case2*(N_estagios-2);

    posicao[gid]=-gid;

    while(posicao[L-1]<N_operacoes_case1+(N_filtragens_decimacao))
    {    
        soma_entrada[gid]=0;
        soma_saida[gid]=0;

        if(posicao[gid]>N_operacoes_case1-1 && posicao[gid]<N_operacoes_case1+(N_filtragens_decimacao)) //Decimacao
        {
            for(int i=N_decimacao; i>0; i--)
            {
                entrada_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]=entrada_decimacao[i-1+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)];
            }
            entrada_decimacao[(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]=amostra_in[gid];

            for(int i=0; i<N_decimacao_mais_1; i++)
            {
                entrada_multiplicada_decimacao[i]=entrada_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao_mais_1)]*b_decimacao[i];
                soma_entrada[gid]+=entrada_multiplicada_decimacao[i];
            }
            for(int i=0; i<N_decimacao; i++)
            {
                saida_multiplicada_decimacao[i]=saida_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao)]*a_decimacao[i];
                soma_saida[gid]+=saida_multiplicada_decimacao[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_decimacao-1; i>0; i--)
            {
                saida_decimacao[i+(posicao[gid]-N_operacoes_case1)*(N_decimacao)]=saida_decimacao[i-1+(posicao[gid]-N_operacoes_case1)*(N_decimacao)];
            }
            saida_decimacao[(posicao[gid]-N_operacoes_case1)*(N_decimacao)]=amostra_in[gid];

            if(gid==0 && posicao[gid]==N_operacoes_case1+(N_filtragens_decimacao)-1)
            {
                amostra_out[0]=amostra_in[gid];
            }

        }
        else if(posicao[gid]>-1 && posicao[gid]<N_operacoes_case1) //Distorcao
        {
            for(int i=N_aliasing; i>0; i--)
            {
                entrada_aliasing[i+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]=entrada_aliasing[i-1+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)];
            }
            entrada_aliasing[(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]=atan(grau_distorcao*amostra_in[gid])/atan(grau_distorcao);

            for(int i=0; i<N_aliasing_mais_1; i++)
            {
                entrada_multiplicada_aliasing[i]=entrada_aliasing[i+(posicao[gid]+offset_decimacao)*(N_aliasing_mais_1)]*b_aliasing[i];
                soma_entrada[gid]+=entrada_multiplicada_aliasing[i];
            }
            for(int i=0; i<N_aliasing; i++)
            {
                saida_multiplicada_aliasing[i]=saida_aliasing[i+(posicao[gid]+offset_decimacao)*N_aliasing]*a_aliasing[i];
                soma_saida[gid]+=saida_multiplicada_aliasing[i];
            }

            amostra_in[gid]=soma_entrada[gid]-soma_saida[gid];

            for(int i=N_aliasing-1; i>0; i--)
            {
                saida_aliasing[i+(posicao[gid]+offset_decimacao)*N_aliasing]=saida_aliasing[i-1+(posicao[gid]+offset_decimacao)*N_aliasing];
            }
            saida_aliasing[(posicao[gid]+offset_decimacao)*N_aliasing]=amostra_in[gid];
        }

        posicao[gid]+=1;

        __syncthreads();
    }
}

//Kernel para verificar uma variavel do device 
__global__ void Verifica_buffer(double *bufferd, int i, int nome)
{
    if(nome == 0)
        printf("H/D buffer[%d] = %e\n", i, bufferd[i]);
    else
        printf("D/H buffer[%d] = %e\n", i, bufferd[i]);
}

//Kernel para verificar um vetor do device
__global__ void Verifica_vetor(double *bufferd, int N)
{
    for (int i=0; i<N;i++)
    {
        printf("vetor[%d]=%e\n", i, bufferd[i]);
    }
}

int main(int argc, char *argv[]){
    if (argc != 2)
    {
        printf("./executavel [N_repeticoes]\n");
        return 1;
    }

    printf("Definindo arquivo de entrada e saída: \n");
    
    //Arquivo de entrada e saida
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Definições do arquivo de leitura e saida
    SNDFILE *file_in,*file_out ;   //Arquivo de entrada e saída
    SF_INFO sfinfo_in,sfinfo_out ; //Arquivo de informações de entrada e saída
    sfinfo_in.format = 0;          //Documentação do libsnd manda fazer isso para arquivos de leitura
    file_in = sf_open ("sweep_48khz_5s.wav", SFM_READ, &sfinfo_in); //Determina o arquivo de entrada
    sfinfo_out=sfinfo_in;
    file_out = sf_open ("delay_atualizado.wav", SFM_WRITE, &sfinfo_out); //Determina o arquivo de saída
    sf_command (file_out, SFC_SET_CLIPPING, NULL, SF_TRUE) ;
    printf("Informações sobre o arquivo de entrada: \n");
    printf("Taxa de amostragem = %d, Frames = %d, Canais = %d \n", (int)sfinfo_in.samplerate, (int)sfinfo_in.frames, (int)sfinfo_in.channels);
    int read_count = 1; //Mostra algumas caratersiticas do arquivo de entrada 
    ////////////////////////////////////////////////////////////////////////////////////////////////

    printf("Alocando memória: \n");

    //Definicao das variaveis e alocacao de memoria
    ////////////////////////////////////////////////////////////////////////////////////////////////
    //Definições do Projeto
    const double grau_distorcao=0.1; //Grau de distorcao do atan
    //const int N_amostras = sfinfo_in.frames; //Numero de amostras do arquivo de entrada
    const int N_amostras = 1000; //Para verficacao da timeline

    //Variaveis da CPU
    double *amostra_in_host; //Amostra para a entrada no processamento feito na GPU
    double *amostra_out_host; //Amostra para a saída no processamento feito na GPU

    //Alocação das Variaveis da CPU
    hipHostAlloc((void**)&amostra_in_host, sizeof(double)*N_amostras,hipHostMallocDefault);
    hipHostAlloc((void**)&amostra_out_host, sizeof(double)*N_amostras,hipHostMallocDefault);

    //Definição dos coeficientes do filtro na CPU
    double a_interpol_host[N_interpol] = {
        #include "a_interpol_30.txt"
    };
    double b_interpol_host[N_interpol_mais_1] = {
        #include "b_interpol_30.txt"
    };
    double a_decimacao_host[N_decimacao] = {
        #include "a_decimacao_30.txt"
    };
    double b_decimacao_host[N_decimacao_mais_1] = {
        #include "b_decimacao_30.txt"
    };
    double a_aliasing_host[N_aliasing] = {
        #include "a_aliasing_30.txt"
    };
    double b_aliasing_host[N_aliasing_mais_1] = {
        #include "b_aliasing_30.txt"
    };

    //Variaveis da GPU
    double *buffer_device_in; //Buffer do devide para leitura do arquivo
    double *buffer_device_out; //Buffer do para escrever amostras processadas
    double *amostra_estagio; //Buffer intermediario das distorcoes
    double *a_interpol; //Vetor dos coeficientes recursivos do filtro de interpolação 
    double *b_interpol; //Vetor dos coeficientes relacionados a entrada do filtro de interpolação
    double *a_decimacao; //Vetor dos coeficientes recursivos do filtro de decimacao
    double *b_decimacao; //Vetor dos coeficientes relacionados a entrada do filtro de decimação
    double *a_aliasing; //Vetor dos coeficientes recursivos do filtro pós distorção 
    double *b_aliasing; //Vetor dos coeficientes relacionados a entrada do filtro pós distorção
    double *entrada_interpol; //Vetor de entrada na interpolação
    double *saida_interpol; //Vetor de saída na interpolação
    double *entrada_decimacao; //Vetor de entrada na decimação
    double *saida_decimacao; //Vetor de saída na decimação
    double *entrada_aliasing; //Vetor de entrada no filtro pós distorção
    double *saida_aliasing; //Vetor de saída no filtro pós distorção


    //Alocação das Variaveis da GPU
    hipMalloc((void**)&amostra_estagio, sizeof(double)* L*N_buffer);
    hipMalloc((void**)&buffer_device_in, sizeof(double)*N_buffer);
    hipMalloc((void**)&buffer_device_out, sizeof(double)*N_buffer);
    hipMalloc((void**)&a_interpol, sizeof(double) * N_interpol);
    hipMalloc((void**)&b_interpol, sizeof(double) * (N_interpol_mais_1));
    hipMalloc((void**)&a_decimacao, sizeof(double) * N_decimacao);
    hipMalloc((void**)&b_decimacao, sizeof(double) * (N_decimacao_mais_1));
    hipMalloc((void**)&a_aliasing, sizeof(double) * N_aliasing);
    hipMalloc((void**)&b_aliasing, sizeof(double) * (N_aliasing_mais_1));
    hipMalloc((void**)&saida_interpol, sizeof(double) * (N_filtragens_interpol)*N_interpol);
    hipMalloc((void**)&entrada_interpol, sizeof(double) * (N_filtragens_interpol)*(N_interpol_mais_1));
    hipMalloc((void**)&saida_decimacao, sizeof(double) * (N_filtragens_decimacao)*N_decimacao);
    hipMalloc((void**)&entrada_decimacao, sizeof(double) * (N_filtragens_decimacao)*(N_decimacao_mais_1));
    hipMalloc((void**)&entrada_aliasing, sizeof(double) * N_distorcao_max*(N_aliasing_mais_1));
    hipMalloc((void**)&saida_aliasing, sizeof(double) * N_distorcao_max*N_aliasing);
    

    //Definição dos coeficientes dos filtros na GPU;
    hipMemcpy(a_interpol, a_interpol_host, sizeof(double)*N_interpol, hipMemcpyHostToDevice);
    hipMemcpy(b_interpol, b_interpol_host, sizeof(double)*(N_interpol_mais_1), hipMemcpyHostToDevice);
    hipMemcpy(a_decimacao, a_decimacao_host, sizeof(double)*N_decimacao, hipMemcpyHostToDevice);
    hipMemcpy(b_decimacao, b_decimacao_host, sizeof(double)*(N_decimacao_mais_1), hipMemcpyHostToDevice);
    hipMemcpy(a_aliasing, a_aliasing_host, sizeof(double)*N_aliasing, hipMemcpyHostToDevice);
    hipMemcpy(b_aliasing, b_aliasing_host, sizeof(double)*(N_aliasing_mais_1), hipMemcpyHostToDevice);


    //Definindo filtros com condicoes iniciais nulas
    hipMemset((void**)&saida_interpol, 0, sizeof(double)*(N_filtragens_interpol)*N_interpol); 
    hipMemset((void**)&entrada_interpol, 0, sizeof(double)*(N_filtragens_interpol)*(N_interpol_mais_1)); 
    hipMemset((void**)&saida_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*N_decimacao);
    hipMemset((void**)&entrada_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*(N_decimacao_mais_1));
    hipMemset((void**)&saida_aliasing, 0, sizeof(double)*N_aliasing*N_distorcao_max);
    hipMemset((void**)&entrada_aliasing, 0, sizeof(double)*(N_aliasing_mais_1)*N_distorcao_max);
    hipMemset((void**)&buffer_device_out, 2, sizeof(double)*N_buffer);
    hipMemset((void**)&buffer_device_in, 0, sizeof(double)*N_buffer);
    
    // for(int i=0; i<sfinfo_in.frames; i++)
    // {
    //     read_count = (int) sf_read_double (file_in, &amostra_in_host[i], 1);
    // }
    sf_read_double (file_in, amostra_in_host, sfinfo_in.frames);

    int N_eventos= (N_max_estagios+1);
    int N_eventos_mais_um = N_eventos+1;
    int N_streams= (N_max_estagios+2);
    int N_estagios_menos_1= (N_max_estagios-1);

    //Definindo Streams
    hipStream_t estagio[N_streams];
    for(int i=0; i<(N_streams); i++)
    {
        hipStreamCreate(&estagio[i]);
    }

    //Definindo Eventos
    hipEvent_t *fim_estagio;
    hipEvent_t *delay_in;
    hipEvent_t *delay_out;
    fim_estagio=(hipEvent_t*)malloc(sizeof(hipEvent_t)* (N_buffer+1)*(N_eventos+1));
    delay_in=(hipEvent_t*)malloc(sizeof(hipEvent_t)* N_amostras);
    delay_out=(hipEvent_t*)malloc(sizeof(hipEvent_t)* N_amostras);

    for(int i=0; i<(N_buffer+1)*(N_eventos+1); i++)
    {
        hipEventCreateWithFlags(&fim_estagio[i],hipEventDisableTiming);
    }
    
     for(int i=0; i<N_amostras; i++)
    {
        hipEventCreate(&delay_in[i]);
        hipEventCreate(&delay_out[i]);
    }
    
    //Variavel de confirmacao de eventos
    int* event_ok;
    event_ok=(int*)calloc((N_buffer+1)*(N_eventos+1),sizeof(int));

    //Variaveis de medida de tempo
    float delay_media;
    float delay;
    float delay_max;

    //Contadores de cada estagio
    int amostras_processadas[N_streams];
    for(int i=0; i<N_streams; i++)
    {
        amostras_processadas[i]=0;
    }
    
    //Arquivo de log
    FILE *fp = fopen("delay_gpu_L=30_5k_cuda_2.csv","w");
    FILE *fp_amostras = fopen("delay_gpu_L=30_5k_amostras_cuda_2.csv","w");
    fprintf(fp,"N_estagios,N_operacoes_case01,N_operacoes_case2,Média Delay_max [ms]\n");
    fprintf(fp_amostras,"N_estagios,N_operacoes_case01,N_operacoes_case2,Média Delay_max [ms]\n");
    
    ////////////////////////////////////////////////////////////////////////////////////////////////
    
    int N_operacoes_case1;
    int N_operacoes_case2;
    double soma_total=0;
    int nro_repeticoes_de_amostra = atoi(argv[1]);

    int contador = 0;
    int pct=0;
    int N_distorcao;
    
    
    for(int N_estagios=2 ; N_estagios<25; N_estagios++)
    {
        if(144%N_estagios==0)
        {
            for(int N_operacoes_case0=(144)/N_estagios; N_operacoes_case0<=(5040)/N_estagios; N_operacoes_case0=N_operacoes_case0+144/N_estagios)
            {   
                delay_media=0;
                soma_total=0;
                N_distorcao = (N_operacoes_case0*(N_estagios));
                N_operacoes_case1 = N_operacoes_case0;
                N_operacoes_case2 = N_operacoes_case0;
                for(int i = 0; i < nro_repeticoes_de_amostra; i++)
                {
                    delay_max=0;
                    int amostras_processadas_totais=0;
                    int vezes_zeradas = 0;
                    int amostras_enviadas = 0;
                    N_eventos= (N_estagios+1);
                    N_eventos_mais_um = N_eventos+1;
                    N_estagios_menos_1= (N_estagios-1);
                    hipMemset((void**)&saida_interpol, 0, sizeof(double)*(N_filtragens_interpol)*N_interpol); 
                    hipMemset((void**)&entrada_interpol, 0, sizeof(double)*(N_filtragens_interpol)*(N_interpol_mais_1)); 
                    hipMemset((void**)&saida_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*N_decimacao);
                    hipMemset((void**)&entrada_decimacao, 0, sizeof(double)*(N_filtragens_decimacao)*(N_decimacao_mais_1));
                    hipMemset((void**)&saida_aliasing, 0, sizeof(double)*N_aliasing*N_distorcao_max);
                    hipMemset((void**)&entrada_aliasing, 0, sizeof(double)*(N_aliasing_mais_1)*N_distorcao_max);
                    hipMemset((void**)&buffer_device_out, 0, sizeof(double)*N_buffer);
                    hipMemset((void**)&buffer_device_in, 0, sizeof(double)*N_buffer);
                    

                    for(int i=0; i<(N_max_estagios+2); i++)
                    {
                        amostras_processadas[i]=0;
                    }

                    for(int i=0; i<(N_buffer+1)*(N_max_estagios+2); i++)
                    {
                        event_ok[i]=0;
                    }

                    while (amostras_processadas_totais< N_amostras)
                    {
                        if(amostras_processadas[0]< N_buffer)
                        {
                            hipStreamSynchronize(estagio[1]);
                            if(amostras_enviadas < N_amostras && amostras_processadas[0]==amostras_processadas[1])
                            {
                                hipEventRecord(delay_in[amostras_enviadas],estagio[0]);
                                hipMemcpyAsync(&buffer_device_in[amostras_processadas[0]], &amostra_in_host[amostras_enviadas], sizeof(double), hipMemcpyHostToDevice,estagio[0]);
                                hipEventRecord(fim_estagio[(N_eventos_mais_um)*amostras_processadas[0]],estagio[0]);
                                event_ok[(N_eventos_mais_um)*amostras_processadas[0]]=1;
                                amostras_processadas[0]++;
                                amostras_enviadas++;
                            } 
                        }


                        if(hipEventQuery(fim_estagio[(amostras_processadas[1])*(N_eventos_mais_um)])==hipSuccess && (event_ok[(amostras_processadas[1])*(N_eventos_mais_um)]==1))
                        {
                            event_ok[(amostras_processadas[1])*(N_eventos_mais_um)]=0;

                            Interpolacao<<<1,L,0,estagio[1]>>>(
                                &buffer_device_in[amostras_processadas[1]],&amostra_estagio[L*amostras_processadas[1]],
                                entrada_interpol,saida_interpol,
                                entrada_aliasing,saida_aliasing,
                                a_interpol, b_interpol,
                                a_aliasing, b_aliasing,
                                grau_distorcao,N_operacoes_case0);
                            hipEventRecord(fim_estagio[1+amostras_processadas[1]*(N_eventos_mais_um)],estagio[1]);
                            event_ok[(N_eventos_mais_um)*amostras_processadas[1]+1]=1;
                            amostras_processadas[1]++;
                        }

                        for(int z=1; z<N_estagios_menos_1;z++)
                        {
                            if(hipEventQuery(fim_estagio[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)])==hipSuccess && event_ok[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)]==1)
                            {
                                event_ok[z+amostras_processadas[(z+1)]*(N_eventos_mais_um)]=0;

                                Distorcao<<<1,L,0,estagio[(z+1)]>>>(
                                    &amostra_estagio[L*amostras_processadas[(z+1)]],&amostra_estagio[L*amostras_processadas[(z+1)]],
                                    entrada_aliasing,saida_aliasing,
                                    a_aliasing, b_aliasing,
                                    grau_distorcao, z, N_operacoes_case0, N_operacoes_case2);
                                hipEventRecord(fim_estagio[z+1+amostras_processadas[(z+1)]*(N_eventos_mais_um)],estagio[(z+1)]);
                                event_ok[z+1+(N_eventos_mais_um)*amostras_processadas[(z+1)]]=1;
                                amostras_processadas[(z+1)]++;
                            }
                        }

                        if(hipEventQuery(fim_estagio[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)])==hipSuccess && event_ok[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]==1)
                        {
                            event_ok[(N_estagios_menos_1)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]=0;

                            Decimacao<<<1,L,0,estagio[(N_estagios)]>>>(
                                &amostra_estagio[L*amostras_processadas[(N_estagios)]],&buffer_device_out[amostras_processadas[(N_estagios)]],
                                entrada_aliasing,saida_aliasing,
                                entrada_decimacao, saida_decimacao,
                                a_aliasing, b_aliasing,
                                a_decimacao, b_decimacao,
                                grau_distorcao, N_operacoes_case0, N_operacoes_case2, N_operacoes_case1, N_estagios);
                            hipEventRecord(fim_estagio[(N_estagios)+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)],estagio[(N_estagios)]);
                            event_ok[N_estagios+amostras_processadas[(N_estagios)]*(N_eventos_mais_um)]=1;
                            amostras_processadas[(N_estagios)]++;
                        }

                        if(hipEventQuery(fim_estagio[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]])==hipSuccess && event_ok[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]==1)
                        {
                            event_ok[(N_estagios)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]=0;
                            hipMemcpyAsync(&amostra_out_host[amostras_processadas_totais], &buffer_device_out[amostras_processadas[(N_eventos)]], sizeof(double), hipMemcpyDeviceToHost,estagio[(N_eventos)]);
                            hipEventRecord(delay_out[amostras_processadas_totais],estagio[N_eventos]);
                            hipEventRecord(fim_estagio[(N_eventos)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]],estagio[(N_eventos)]);
                            event_ok[(N_eventos)+(N_eventos_mais_um)*amostras_processadas[(N_eventos)]]=1;
                            amostras_processadas[(N_eventos)]++;
                            amostras_processadas_totais++;
                        }

                        if(amostras_processadas[(N_eventos)] == (N_buffer) && amostras_processadas[0]== (N_buffer) )
                        {
                            if(hipEventQuery(fim_estagio[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)])==hipSuccess && event_ok[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)]==1)
                            {
                                event_ok[(N_eventos)+(N_eventos_mais_um)*(amostras_processadas[(N_eventos)]-1)]=0;
                                for(int k=0; k<N_eventos_mais_um;k++)
                                {
                                    amostras_processadas[k]=0;
                                }
                            }
                        }

                    }

                    hipDeviceSynchronize();

                    for(int z=0; z<N_amostras; z++)
                    {
                        hipEventElapsedTime(&delay,delay_in[z],delay_out[z]);
                        delay_media+=delay;
                        fprintf(fp_amostras,"%d,%d,%d,%f\n", N_estagios, N_operacoes_case0, N_operacoes_case2, delay);
                        if(delay>delay_max)
                        {
                            delay_max=delay;
                        }
                    }

                    soma_total+=delay_max;
                    //printf("N_estagios: %d N_operacoes_case0: %d N_operacoes_case2: %d Delay_max [ms]: %f\n", N_estagios, N_operacoes_case0, N_operacoes_case2, delay_max);
                }
                contador++;

                delay_media=delay_media/(N_amostras*nro_repeticoes_de_amostra);
                // soma_total=soma_total/(nro_repeticoes_de_amostra);
                fprintf(fp,"%d,%d,%d,%f\n", N_estagios, N_operacoes_case0, N_operacoes_case2, delay_media);
                printf("N_estagios: %d N_operacoes_case0: %d N_operacoes_case2: %d Delay_médio [ms]: %f\n", N_estagios, N_operacoes_case0, N_operacoes_case2, delay_media);
            
            }
        }
    }
    ////////////////////////////////////////////////////////////////////////////////////////////////


   
    //Liberando a memória alocada
    ////////////////////////////////////////////////////////////////////////////////////////////////
    free(event_ok);

    
    hipFree(amostra_in_host);
    hipFree(amostra_out_host);

    hipFree(buffer_device_in);
    hipFree(buffer_device_out);
    hipFree(amostra_estagio);

    hipFree(a_interpol);
    hipFree(b_interpol);
    hipFree(a_decimacao);
    hipFree(b_decimacao);
    hipFree(a_aliasing);
    hipFree(b_aliasing);
    hipFree(entrada_interpol);
    hipFree(saida_interpol);
    hipFree(entrada_decimacao);
    hipFree(saida_decimacao);
    hipFree(entrada_aliasing);
    hipFree(saida_aliasing);

    N_eventos = (N_max_estagios+1);

    for(int i=0; i<(N_buffer+1)*(N_eventos+1); i++)
    {
        hipEventDestroy(fim_estagio[i]);
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////

     printf("Fim: \n");
 }
